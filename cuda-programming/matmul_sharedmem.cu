
#include <hip/hip_runtime.h>
#include <stdio.h>
// import more
#include <stdlib.h> 
#include <time.h>

// tile size
#define TILE_SIZE 32

// matmul with shared mem
__global__ void matrixMultiplicationShared(int *A, int *B, int *C, int width) {
    __shared__ int sharedA[TILE_SIZE][TILE_SIZE];
    __shared__ int sharedB[TILE_SIZE][TILE_SIZE];
    
    // get thread indices
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    
    int sum = 0;
    
    // loop over the tile
    for (int t = 0; t < (width + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // Load data into shared memory
        if (row < width && t * TILE_SIZE + tx < width) {
            sharedA[ty][tx] = A[row * width + t * TILE_SIZE + tx];
        } else {
            sharedA[ty][tx] = 0;
        }
        
        if (t * TILE_SIZE + ty < width && col < width) {
            sharedB[ty][tx] = B[(t * TILE_SIZE + ty) * width + col];
        } else {
            sharedB[ty][tx] = 0;
        }
        
        // sync to ensure all data is there before compute
        __syncthreads();
        
        // Compute partial sum for this tile
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += sharedA[ty][k] * sharedB[k][tx];
        }
        
        // sync to make sure this tile complete
        __syncthreads();
    }
    
    // get the final result
    if (row < width && col < width) {
        C[row * width + col] = sum;
    }
}

__global__ void matrixMultiplication(int *A, int *B, int *C, int width) {
    // TODO : Implement matrix multiplication kernel
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width) {
        int single_value = 0;
        for (int i = 0; i < width; i++) {
            single_value += A[row * width + i] * B[i * width + col]; // sum up all the values
        }
        // assign that single value to C entries
        C[row * width + col] = single_value;
    }
}

// cpu for matrix mul 
void cpuMatrixMul(int* A, int* B, int* C, int width) {
    for (int row = 0; row < width; row++) {
        for (int col = 0; col < width; col++) {
            int single_value = 0;
            for (int i = 0; i < width; i++) {
                single_value += A[row * width + i] * B[i * width + col];
            }            
            C[row * width + col] = single_value;
        }
    }
}

int main() {
    const int width = 512; // Matrix width
    int A[width * width], B[width * width], C[width * width], D[width * width]; // Host matrices
    int *d_A, *d_B, *d_C; // device pointers
    int iterations = 10; // iterations for timing
    
    // size for the vectors
    size_t size = width * width * sizeof(int);
    
    printf("Using matrix size = %d with width = %d\n", width*width, width);
    printf("Using shared memory tile size = %d x %d\n", TILE_SIZE, TILE_SIZE);
    
    // TODO : Initialize matrices 'A' and 'B' with random values
    srand(time(NULL));
    for (int i = 0; i < width * width; i++) {
        // rand between 1 and 10: https://stackoverflow.com/questions/17846212/generate-a-random-number-between-1-and-10-in-c
        A[i] = rand() % 10;
        B[i] = rand() % 10;
    }

    // timing stuff
    hipEvent_t gpu_start, gpu_stop;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);


    // SHARED MEM HERE
    float totalSharedGpuTime = 0.0f;
    
    // define multiple iterations and avg 
    for (int i = 0; i < iterations; i++) {
        // TODO : Allocate device memory for matrices 'A ', 'B ', and 'C '
        // cudaMalloc(void **pointer, size_t nbytes) 
        hipMalloc((void**)&d_A, size);
        hipMalloc((void**)&d_B, size);
        hipMalloc((void**)&d_C, size);

        // TODO : Copy matrices 'A' and 'B' from host to device 
        // cudaMemcpy(void *dst, void *src, size_t nbytes, enum cudaMemcpyKind direction);
        hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

        // Define grid and block dimensions for shared memory kernel
        dim3 dimBlock(TILE_SIZE, TILE_SIZE);
        dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (width + dimBlock.y - 1) / dimBlock.y);

        // start timing
        hipEventRecord(gpu_start, 0);
        
        // Launch kernel with shared memory
        matrixMultiplicationShared<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, width);
        
        // TODO : Copy the result matrix 'C ' from device to host
        hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
        
        hipEventRecord(gpu_stop, 0);
        hipEventSynchronize(gpu_stop);

        float iterationTime = 0.0f;
        hipEventElapsedTime(&iterationTime, gpu_start, gpu_stop);
        totalSharedGpuTime += iterationTime;

        // TODO : What is needed here ? -> cudaFree
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }

    // get the avg gpu time for shared mem
    float avgSharedGpuTime = totalSharedGpuTime / iterations;


    // REGULAR HERE
    float totalRegularGpuTime = 0.0f;
    
    for (int i = 0; i < iterations; i++) {
        // TODO : Allocate device memory for matrices 'A ', 'B ', and 'C '
        // cudaMalloc(void **pointer, size_t nbytes) 
        hipMalloc((void**)&d_A, size);
        hipMalloc((void**)&d_B, size);
        hipMalloc((void**)&d_C, size);

        // TODO : Copy matrices 'A' and 'B' from host to device 
        // cudaMemcpy(void *dst, void *src, size_t nbytes, enum cudaMemcpyKind direction);
        hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

        // Define grid and block dimensions (this is full dim so we dont need to grid-stride stuff)
        dim3 dimGrid((width + 15) / 16, (width + 15) / 16, 1);
        dim3 dimBlock(16, 16, 1);

        // start timing
        hipEventRecord(gpu_start, 0);
        
        // Launch the matrix multiplication kernel
        matrixMultiplication<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, width);

        // TODO : Copy the result matrix 'C ' from device to host
        hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
        
        hipEventRecord(gpu_stop, 0);
        hipEventSynchronize(gpu_stop);

        float iterationTime = 0.0f;
        hipEventElapsedTime(&iterationTime, gpu_start, gpu_stop);
        totalRegularGpuTime += iterationTime;

        // TODO : What is needed here ? -> cudaFree
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }

    // get the avg gpu time for regular
    float avgRegularGpuTime = totalRegularGpuTime / iterations;

    // timings for cpu after 100 iterations
    float totalCpuTime = 0.0f;

    for (int i = 0; i < iterations; i++) {
        clock_t cpu_start = clock();

        cpuMatrixMul(A, B, D, width);

        clock_t cpu_end = clock();
        float iterationTime = 1000.0f * (cpu_end - cpu_start) / CLOCKS_PER_SEC; // convert to ms by x1000
        totalCpuTime += iterationTime;
    }

    // avg cpu time
    float avgCpuTime = totalCpuTime / iterations;

    // Print out the timings for both cpu and gpu (forammted by GPT4o)
    printf("\n===== Performance Comparison =====\n");
    printf("CPU takes %fms\n", avgCpuTime);
    printf("GPU Regular takes %fms\n", avgRegularGpuTime);
    printf("GPU Shared Memory takes %fms\n", avgSharedGpuTime);
    printf("Speedup (Shared vs Regular): %.2fx\n", avgRegularGpuTime / avgSharedGpuTime);
    printf("Speedup (Shared vs CPU): %.2fx\n", avgCpuTime / avgSharedGpuTime);

    // TODO: Verify the shared memory implementation
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);
    
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
    dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (width + dimBlock.y - 1) / dimBlock.y);
    
    matrixMultiplicationShared<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, width);
    
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // TODO : Verify the correctness of the result
    // loop thru each element in each
    int correct = 1;
    for (int i = 0; i < width * width; i++) {
        if (C[i] != D[i]) {
            correct = 0;
            printf("Index mismatch at %d, GPU: %d, CPU: %d\n", i, C[i], D[i]);
            break;
        }
    }

    if (correct == 1) {
        printf("The matrix mul is right as both CPU and GPU matches\n");
    }

    // destroy the cudaEvent stuff
    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);

    return 0;
}
